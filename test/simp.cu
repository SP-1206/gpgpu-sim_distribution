
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addOne(int* arr, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        arr[idx] += 1;
    }
}

int main() {
    const int size = 5000;
    int arr[size];
    int* d_arr;

    // Allocate memory on device
    hipMalloc(&d_arr, sizeof(int) * size);

    // Initialize array values
    for (int i = 0; i < size; i++) {
        arr[i] = i + 1;
    }

    // Copy array to device
    hipMemcpy(d_arr, arr, sizeof(int) * size, hipMemcpyHostToDevice);

    // Launch kernel to add 1 to array values
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addOne<<<blocksPerGrid, threadsPerBlock>>>(d_arr, size);

    // Copy array back to host
    hipMemcpy(arr, d_arr, sizeof(int) * size, hipMemcpyDeviceToHost);

    // Print array values
    for (int i = 0; i < size; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    // Free memory on device
    hipFree(d_arr);

    return 0;
}

